
#include <hip/hip_runtime.h>

#define ALPHA		 0.007
#define DT			 0.2

#define max(x,y)    ((x>y) ? x : y )
#define min(x,y)    ((x<y) ? x : y )

__global__ void updatephi( float *d_phi, float *d_phi1, float *d_D, int imageW, int imageH)

{
	int c= blockIdx.x * blockDim.x + threadIdx.x;
	int r= blockIdx.y * blockDim.y + threadIdx.y;
	int ind= r*imageW+c;

	if(ind<imageW*imageH){

		float dx,dxplus,dxminus,dxplusy,dxminusy;

		float dy, dyplus,dyminus,dyplusx,dyminusx;

		float gradphimax, gradphimin, nplusx, nplusy, nminusx, nminusy, curvature;
		float F, gradphi;

		if(c==0||c==imageW-1){dx=0;} else {dx=(d_phi1[ind+1]-d_phi1[ind-1])/2;}
		if(c==imageW-1){dxplus=0;} else {dxplus=(d_phi1[ind+1]-d_phi1[ind]);}
		if(c==0){dxminus=0;} else {dxminus=(d_phi1[ind]-d_phi1[ind-1]);}
		if(r==0||c==0||c==imageW-1){dxplusy=0;} else {dxplusy=(d_phi1[ind-imageW+1]-d_phi1[ind-imageW-1])/2;}
		if(r==imageH-1||c==0||c==imageW-1){dxminusy=0;} else {dxminusy=(d_phi1[ind+imageW+1]-d_phi1[ind+imageW-1])/2;}

		if(r==0||r==imageH-1){dy=0;} else {dy=(d_phi1[ind-imageW]-d_phi1[ind+imageW])/2;}
		if(r==0){dyplus=0;} else {dyplus=(d_phi1[ind-imageW]-d_phi1[ind]);}
		if(r==imageH-1){dyminus=0;} else {dyminus=(d_phi1[ind]-d_phi1[ind+imageW]);}
		if(r==0||c==imageW-1||r==imageH-1){dyplusx=0;} else {dyplusx=(d_phi1[ind-imageW+1]-d_phi1[ind+imageW+1])/2;}
		if(r==0||c==0||r==imageH-1){dyminusx=0;} else {dyminusx=(d_phi1[ind-imageW-1]-d_phi1[ind+imageW-1])/2;}

		gradphimax=sqrt((sqrt(max(dxplus,0)*max(dxplus,0)+max(-dxminus,0)*max(-dxminus,0)))*(sqrt(max(dxplus,0)*max(dxplus,0)+max(-dxminus,0)*max(-dxminus,0)))
					   +(sqrt(max(dyplus,0)*max(dyplus,0)+max(-dyminus,0)*max(-dyminus,0)))*(sqrt(max(dyplus,0)*max(dyplus,0)+max(-dyminus,0)*max(-dyminus,0))));
		
		gradphimin=sqrt((sqrt(min(dxplus,0)*min(dxplus,0)+min(-dxminus,0)*min(-dxminus,0)))*(sqrt(min(dxplus,0)*min(dxplus,0)+min(-dxminus,0)*min(-dxminus,0)))
					   +(sqrt(min(dyplus,0)*min(dyplus,0)+min(-dyminus,0)*min(-dyminus,0)))*(sqrt(min(dyplus,0)*min(dyplus,0)+min(-dyminus,0)*min(-dyminus,0))));
		
		nplusx= dxplus / sqrt(1.192092896e-07F + (dxplus*dxplus) + ((dyplusx + dy)*(dyplusx + dy)*0.25) );
		nplusy= dyplus / sqrt(1.192092896e-07F + (dyplus*dyplus) + ((dxplusy + dx)*(dxplusy + dx)*0.25) );
		nminusx= dxminus / sqrt(1.192092896e-07F + (dxminus*dxminus) + ((dyminusx + dy)*(dyminusx + dy)*0.25) );
		nminusy= dyminus / sqrt(1.192092896e-07F + (dyminus*dyminus) + ((dxminusy + dx)*(dxminusy + dx)*0.25) );
		curvature= ((nplusx-nminusx)+(nplusy-nminusy))/2;
		
		F = (-ALPHA * d_D[ind]) + ((1-ALPHA) * curvature);
		if(F>0) {gradphi=gradphimax;} else {gradphi=gradphimin;}
		d_phi[ind]=d_phi1[ind] + (DT * F * gradphi);
	
}
}
		



