#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <float.h>
#include <GL/glew.h>
#include <hip/hip_runtime.h>
#include <cutil.h>
#include <math.h>
#include <GL/glut.h>

#define IMAGE			"liver.bmp"

#define ITERATIONS   5000
#define THRESHOLD	 180
#define EPSILON		 40

#define RITS		 50

#define BLOCKDIM_X	 16
#define BLOCKDIM_Y	 32

float *phi, *D;
uchar4 *h_Src, *h_Mask;
int imageW, imageH, N;

float *d_phi, *d_D;
float *d_phi1;

void LoadBMPFile(uchar4 **dst, int *width, int *height, const char *name);
void sedt2d(int *_d,unsigned char *_bimg,int _h,int _w);


int its=0;
unsigned int Timer = 0;
unsigned int ReInitTimer = 0;

int r;
int c;
int i;

__global__ void updatephi( float *d_phi, float *d_phi1, float *d_D, int imageW, int imageH);

void init_phi(){

	int *init;
	unsigned char *mask;
	const char *mask_path = "mask.bmp";
	if((init=(int *)malloc(imageW*imageH*sizeof(int)))==NULL)printf("ME_INIT\n");
	if((phi=(float *)malloc(imageW*imageH*sizeof(float)))==NULL)printf("ME_PHI\n");
	mask = (unsigned char *)malloc(imageW*imageH*sizeof(unsigned char));

	//printf("Init Mask\n");
	LoadBMPFile(&h_Mask, &imageW, &imageH, mask_path);
	

	for(r=0;r<imageH;r++){
		for(c=0;c<imageW;c++){
			mask[r*imageW+c] = (h_Mask[r*imageW+c].x)/255;
			//printf("%3d ", mask[r*imageW+c]);
		}
		//printf("\n");
	}

	sedt2d(init,mask,imageH,imageW);

	//printf("sdf of init mask\n");
	for(r=0;r<imageH;r++){
		for(c=0;c<imageW;c++){
			phi[r*imageW+c]=(float)init[r*imageW+c];
			if(phi[r*imageW+c]>0){
				phi[r*imageW+c]=0.5*sqrt(abs(phi[r*imageW+c]));
			} else {
				phi[r*imageW+c]=-0.5*sqrt(abs(phi[r*imageW+c]));
			}
			//printf("%6.3f ", phi[r*imageW+c]);
		}
		//printf("\n");
	}

	free(init);
	free(mask);
}

void reinit_phi(){

	int *intphi;
	unsigned char *reinit;
	if((intphi=(int *)malloc(imageW*imageH*sizeof(int)))==NULL)printf("ME_INIT\n");
	reinit=(unsigned char *)malloc(imageW*imageH*sizeof(unsigned char));//TODO check

	for(i=0;i<N;i++){
		if(phi[i]<0){
			phi[i]=1;
		} else {
			phi[i]=0;
		}
		reinit[i]=(int)phi[i];
	}


	sedt2d(intphi,reinit,imageH,imageW);

	/*printf("ReInit @ %4d its\n",its);*/
	for(r=0;r<imageH;r++){
		for(c=0;c<imageW;c++){
			phi[r*imageW+c]=(float)intphi[r*imageW+c];
			if(phi[r*imageW+c]>0){
				phi[r*imageW+c]=0.5*sqrt(abs(phi[r*imageW+c]));
			} else {
				phi[r*imageW+c]=-0.5*sqrt(abs(phi[r*imageW+c]));
			}
			//printf("%6.3f ", phi[r*imageW+c]);
		}
		//printf("\n");
	}

	free(reinit);
	free(intphi);
}

void cuda_update(){


	dim3 dimGrid( ((imageW-1)/BLOCKDIM_X) + 1, ((imageH-1)/BLOCKDIM_Y) +1 );
	dim3 dimBlock(BLOCKDIM_X, BLOCKDIM_Y);

	
	updatephi<<< dimGrid, dimBlock>>>(d_phi, d_phi1, d_D,  imageW, imageH);

	
	d_phi1=d_phi;
	


}

void disp(void){
	
	
	glClear(GL_COLOR_BUFFER_BIT);

	
	cuda_update();
	

	its++;

	if(its<ITERATIONS){
		glutPostRedisplay();
		
		if(its%50==0){
			
			printf("Iteration %3d Total Time: %3.2f ReInit Time: %3.2f\n", its, 0.001*cutGetTimerValue(Timer), 0.001*cutGetTimerValue(ReInitTimer));
			
			cutStartTimer(ReInitTimer); // ReInit Timer Start
			hipMemcpy(phi, d_phi, sizeof(float)*imageW*imageH, hipMemcpyDeviceToHost);

			reinit_phi(); // ReInit

			glDrawPixels(imageW, imageH, GL_GREEN, GL_FLOAT, phi);
			glutSwapBuffers();
			cutStopTimer(ReInitTimer); // ReInit Timer Stop
		}

	} else {
		
		printf("Iteration %3d Total Time: %3.2f ReInit Time: %3.2f\n", its, 0.001*cutGetTimerValue(Timer), 0.001*cutGetTimerValue(ReInitTimer));

		hipMemcpy(phi, d_phi, sizeof(float)*imageW*imageH, hipMemcpyDeviceToHost);
		glDrawPixels(imageW, imageH, GL_GREEN, GL_FLOAT, phi);
		glutSwapBuffers();



	}
	
}

int main(int argc, char** argv){

	// Load the Input Image using BMPLoader
	const char *image_path = IMAGE;
	LoadBMPFile(&h_Src, &imageW, &imageH, image_path);
	D = (float *)malloc(imageW*imageH*sizeof(float));
	for(r=0;r<imageH;r++){
		for(c=0;c<imageW;c++){
			D[r*imageW+c] = h_Src[r*imageW+c].x;
		}
	}

	N = imageW*imageH;

	// Threshold based on hash defined paramters
	for(i=0;i<N;i++){
		D[i] = EPSILON - abs(D[i] - THRESHOLD);
	}

	// Init phi to SDF
	init_phi();

	// Set up CUDA Timer
	cutCreateTimer(&Timer);
	cutCreateTimer(&ReInitTimer);

	cutStartTimer(Timer);

	// Allocate Memory on Device
	hipMalloc((void**)&d_D,        sizeof(float)*imageW*imageH);
	hipMalloc((void**)&d_phi,      sizeof(float)*imageW*imageH);
	hipMalloc((void**)&d_phi1,         sizeof(float)*imageW*imageH);

	// Copy Host Thresholding Data to Device Memory
	hipMemcpy(d_D, D,				sizeof(float)*imageW*imageH, hipMemcpyHostToDevice);
	hipMemcpy(d_phi1, phi, sizeof(float)*imageW*imageH, hipMemcpyHostToDevice);

	// Init GL Window
	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_ALPHA | GLUT_DOUBLE);
	glutInitWindowSize(imageW,imageH);
	glutInitWindowPosition(100,100);
	glutCreateWindow("GL Level Set Evolution");
	glClearColor(0.0,0.0,0.0,0.0);
	glutDisplayFunc(disp);
	glutMainLoop();

		hipFree(d_D);
		hipFree(d_phi1);
		hipFree(d_phi);
}




//TODO Memory Malloc Free

//TODO Comment Code
