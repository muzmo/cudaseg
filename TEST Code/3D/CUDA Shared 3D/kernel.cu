#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cutil.h>

#define DT			 0.1

#define max(x,y)    ((x>y) ? x : y )
#define min(x,y)    ((x<y) ? x : y )

#define INDEX(i,j,j_off)  (i +__mul24(j,j_off))

#define BLOCKDIM_X	 32
#define BLOCKDIM_Y	 4
#define BLOCKDIM_Z	 1

__global__ void updatephi( float *d_phi, float *d_phi1, float *d_D, int NX, int NY, int NZ, float alpha, int pitch)
{

	float dx,dy,dz;
	float dxplus, dyplus, dzplus, dxminus, dyminus, dzminus;
	float dxplusy, dxminusy, dxplusz, dxminusz, dyplusx, dyminusx, dyplusz, dyminusz, dzplusx, dzminusx, dzplusy, dzminusy;

	float gradphimax, gradphimin, nplusx, nplusy, nplusz, nminusx, nminusy, nminusz, curvature;
	float F, gradphi;

	int   indg, indg_h, indg0;
	int   i, j, k, ind, ind_h, halo, active;

	#define IOFF  1
	#define JOFF  (BLOCKDIM_X+2)
	#define KOFF  (BLOCKDIM_X+2)*(BLOCKDIM_Y+2)

	int NXM1 = NX-1;
	int NYM1 = NY-1;
	int NZM1 = NZ-1;

	__shared__ float s_data[3*(BLOCKDIM_X+2)*(BLOCKDIM_Y+2)];

	k    =  threadIdx.y*BLOCKDIM_X + threadIdx.x;
	halo = k < 2*(BLOCKDIM_X+BLOCKDIM_Y+2);

	if (halo) {
		if (threadIdx.y<2) {               // y-halos (coalesced)
			i = threadIdx.x;
			j = threadIdx.y*(BLOCKDIM_Y+1) - 1;
		}
		else {                             // x-halos (not coalesced)
			i = (k%2)*(BLOCKDIM_X+1) - 1;
			j =  k/2 - BLOCKDIM_X - 1;
		}

		ind_h  = INDEX(i+1,j+1,BLOCKDIM_X+2);

		i      = INDEX(i,blockIdx.x,BLOCKDIM_X);   // global indices
		j      = INDEX(j,blockIdx.y,BLOCKDIM_Y);
		indg_h = INDEX(i,j,pitch);

		halo   =  (i>=0) && (i<NX) && (j>=0) && (j<NY);
	}

	//
	// then set up indices for main block
	//

	i    = threadIdx.x;
	j    = threadIdx.y;
	ind  = INDEX(i+1,j+1,BLOCKDIM_X+2) ;

	i    = INDEX(i,blockIdx.x,BLOCKDIM_X);     // global indices
	j    = INDEX(j,blockIdx.y,BLOCKDIM_Y);
	indg = INDEX(i,j,pitch);

	active = (i<NX) && (j<NY);

	//
	// read initial plane of u1 array
	//

	if (active) s_data[ind+KOFF+KOFF] = d_phi1[indg];
	if (halo) s_data[ind_h+KOFF+KOFF] = d_phi1[indg_h];

	

	for(int k=0;k<NZ;k++){

		if (active) {
			indg0 = indg;
			indg  = INDEX(indg,NY,pitch);
			s_data[ind-KOFF+KOFF] = s_data[ind+KOFF];
			s_data[ind+KOFF]      = s_data[ind+KOFF+KOFF];
			if (k<NZ-1)
				s_data[ind+KOFF+KOFF] = d_phi1[indg];
		}

		if (halo) {
			indg_h = INDEX(indg_h,NY,pitch);
			s_data[ind_h-KOFF+KOFF] = s_data[ind_h+KOFF];
			s_data[ind_h+KOFF]      = s_data[ind_h+KOFF+KOFF];
			if (k<NZ-1)
				s_data[ind_h+KOFF+KOFF] = d_phi1[indg_h];
		}

		if (active) {
			
			int ind2=ind+KOFF;

			if(i==0||i==NXM1){dx=0;} else {dx=(s_data[ind2+IOFF]-s_data[ind2-IOFF])/2;}
			if(j==0||j==NYM1){dy=0;} else {dy=(s_data[ind2-JOFF]-s_data[ind2+JOFF])/2;}
			if(k==0||k==NZM1){dz=0;} else {dz=(s_data[ind2+KOFF]-s_data[ind2-KOFF])/2;}

			if(i==NXM1){dxplus=0;}   else {dxplus =(s_data[ind2+IOFF]-s_data[ind2     ]);}
			if(j==0){dyplus=0;}		 else {dyplus =(s_data[ind2-JOFF]-s_data[ind2     ]);}
			if(k==NZM1){dzplus=0;}   else {dzplus =(s_data[ind2+KOFF]-s_data[ind2     ]);}
			if(i==0){dxminus=0;}     else {dxminus=(s_data[ind2     ]-s_data[ind2-IOFF]);}
			if(j==NYM1){dyminus=0;}  else {dyminus=(s_data[ind2     ]-s_data[ind2+JOFF]);}
			if(k==0){dzminus=0;}     else {dzminus=(s_data[ind2     ]-s_data[ind2-KOFF]);}

			if(i==0||i==NXM1||j==0){dxplusy=0;}		 else {dxplusy =(s_data[ind2-JOFF+IOFF]-s_data[ind2-JOFF-IOFF])/2;}
			if(i==0||i==NXM1||j==NYM1){dxminusy=0;}  else {dxminusy=(s_data[ind2+JOFF+IOFF]-s_data[ind2+JOFF-IOFF])/2;}
			if(i==0||i==NXM1||k==NZM1) {dxplusz=0;}  else {dxplusz =(s_data[ind2+KOFF+IOFF]-s_data[ind2+KOFF-IOFF])/2;}
			if(i==0||i==NXM1||k==0) {dxminusz=0;}	 else {dxminusz=(s_data[ind2-KOFF+IOFF]-s_data[ind2-KOFF-IOFF])/2;}
			if(j==0||j==NYM1||i==NXM1){dyplusx=0;}   else {dyplusx =(s_data[ind2-JOFF+IOFF]-s_data[ind2+JOFF+IOFF])/2;}
			if(j==0||j==NYM1||i==0){dyminusx=0;}	 else {dyminusx=(s_data[ind2-JOFF-IOFF]-s_data[ind2+JOFF-IOFF])/2;}
			if(j==0||j==NYM1||k==NZM1) {dyplusz=0;}  else {dyplusz =(s_data[ind2+KOFF-JOFF]-s_data[ind2+KOFF+JOFF])/2;}
			if(j==0||j==NYM1||k==0) {dyminusz=0;}	 else {dyminusz=(s_data[ind2-KOFF-JOFF]-s_data[ind2-KOFF+JOFF])/2;}
			if(k==0||k==NZM1||i==NXM1) {dzplusx=0;}  else {dzplusx =(s_data[ind2+IOFF+KOFF]-s_data[ind2+IOFF-KOFF])/2;}
			if(k==0||k==NZM1||i==0) {dzminusx=0;}	 else {dzminusx=(s_data[ind2-IOFF+KOFF]-s_data[ind2-IOFF-KOFF])/2;}
			if(k==0||k==NZM1||j==0) {dzplusy=0;}	 else {dzplusy =(s_data[ind2-JOFF+KOFF]-s_data[ind2-JOFF-KOFF])/2;}
			if(k==0||k==NZM1||j==NYM1) {dzminusy=0;} else {dzminusy=(s_data[ind2+JOFF+KOFF]-s_data[ind2+JOFF-KOFF])/2;}


			gradphimax=sqrt((sqrt(max(dxplus,0)*max(dxplus,0)+max(-dxminus,0)*max(-dxminus,0)))*(sqrt(max(dxplus,0)*max(dxplus,0)+max(-dxminus,0)*max(-dxminus,0)))
				+(sqrt(max(dyplus,0)*max(dyplus,0)+max(-dyminus,0)*max(-dyminus,0)))*(sqrt(max(dyplus,0)*max(dyplus,0)+max(-dyminus,0)*max(-dyminus,0)))
				+(sqrt(max(dzplus,0)*max(dzplus,0)+max(-dzminus,0)*max(-dzminus,0)))*(sqrt(max(dzplus,0)*max(dzplus,0)+max(-dzminus,0)*max(-dzminus,0))));

			gradphimin=sqrt((sqrt(min(dxplus,0)*min(dxplus,0)+min(-dxminus,0)*min(-dxminus,0)))*(sqrt(min(dxplus,0)*min(dxplus,0)+min(-dxminus,0)*min(-dxminus,0)))
				+(sqrt(min(dyplus,0)*min(dyplus,0)+min(-dyminus,0)*min(-dyminus,0)))*(sqrt(min(dyplus,0)*min(dyplus,0)+min(-dyminus,0)*min(-dyminus,0)))
				+(sqrt(min(dzplus,0)*min(dzplus,0)+min(-dzminus,0)*min(-dzminus,0)))*(sqrt(min(dzplus,0)*min(dzplus,0)+min(-dzminus,0)*min(-dzminus,0))));

			nplusx = dxplus / sqrt(1.192092896e-07F + (dxplus*dxplus) + ((dyplusx + dy)*(dyplusx + dy)*0.25) + ((dzplusx + dz)*(dzplusx + dz)*0.25));
			nplusy = dyplus / sqrt(1.192092896e-07F + (dyplus*dyplus) + ((dxplusy + dx)*(dxplusy + dx)*0.25) + ((dzplusy + dz)*(dzplusy + dz)*0.25));
			nplusz = dzplus / sqrt(1.192092896e-07F + (dzplus*dzplus) + ((dxplusz + dz)*(dxplusz + dz)*0.25) + ((dyplusz + dy)*(dyplusz + dy)*0.25));

			nminusx=dxminus / sqrt(1.192092896e-07F + (dxminus*dxminus) + ((dyminusx + dy)*(dyminusx + dy)*0.25) + ((dzminusx + dz)*(dzminusx + dz)*0.25));
			nminusy=dyminus / sqrt(1.192092896e-07F + (dyminus*dyminus) + ((dxminusy + dx)*(dxminusy + dx)*0.25) + ((dzminusy + dz)*(dzminusy + dz)*0.25));
			nminusz=dzminus / sqrt(1.192092896e-07F + (dzminus*dzminus) + ((dxminusz + dz)*(dxminusz + dz)*0.25) + ((dyminusz + dy)*(dyminusz + dy)*0.25));

			curvature= ((nplusx-nminusx)+(nplusy-nminusy)+(nplusz-nminusz))/2;

			F = (-alpha * d_D[indg0]) + ((1-alpha) * curvature);
			if(F>0) {gradphi=gradphimax;} else {gradphi=gradphimin;}
			d_phi[indg0]=s_data[ind2] + (DT * F * gradphi);
		}

		__syncthreads();

	}

}


