#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <float.h>
#include <GL/glew.h>
#include <hip/hip_runtime.h>
#include <cutil.h>
#include <math.h>
#include <GL/glut.h>

#define IMAGE			"mask.bmp"

#define ITERATIONS   5000
#define THRESHOLD	 180
#define EPSILON		 45

#define RITS		 50

#define BLOCKDIM_X	 32
#define BLOCKDIM_Y	 8


float *phi, *D;
uchar4 *h_Src, *h_Mask;
int imageW, imageH, N, pitch;
size_t pitchbytes;

float *d_phi, *d_D;
float *d_phi1;

void LoadBMPFile(uchar4 **dst, int *width, int *height, const char *name);
void sedt2d(int *_d,unsigned char *_bimg,int _h,int _w);


int its=0;
unsigned int Timer = 0;
unsigned int ReInitTimer = 0;

int r;
int c;
int i;

__global__ void updatephi( float *d_phi, float *d_phi1, float *d_D, int imageW, int imageH);

void init_phi(){

	int *init;
	unsigned char *mask;
	const char *mask_path = "mask.bmp";
	if((init=(int *)malloc(imageW*imageH*sizeof(int)))==NULL)printf("ME_INIT\n");
	if((phi=(float *)malloc(imageW*imageH*sizeof(float)))==NULL)printf("ME_PHI\n");
	mask = (unsigned char *)malloc(imageW*imageH*sizeof(unsigned char));

	//printf("Init Mask\n");
	LoadBMPFile(&h_Mask, &imageW, &imageH, mask_path);
	

	for(r=0;r<imageH;r++){
		for(c=0;c<imageW;c++){
			mask[r*imageW+c] = (h_Mask[r*imageW+c].x)/255;
			//printf("%3d ", mask[r*imageW+c]);
		}
		//printf("\n");
	}

	sedt2d(init,mask,imageH,imageW);

	//printf("sdf of init mask\n");
	for(r=0;r<imageH;r++){
		for(c=0;c<imageW;c++){
			phi[r*imageW+c]=(float)init[r*imageW+c];
			if(phi[r*imageW+c]>0){
				phi[r*imageW+c]=0.5*sqrt(abs(phi[r*imageW+c]));
			} else {
				phi[r*imageW+c]=-0.5*sqrt(abs(phi[r*imageW+c]));
			}
			//printf("%6.3f ", phi[r*imageW+c]);
		}
		//printf("\n");
	}

	free(init);
	free(mask);
}

void reinit_phi(){

	int *intphi;
	unsigned char *reinit;
	if((intphi=(int *)malloc(imageW*imageH*sizeof(int)))==NULL)printf("ME_INIT\n");
	reinit=(unsigned char *)malloc(imageW*imageH*sizeof(unsigned char));//TODO check

	for(i=0;i<N;i++){
		if(phi[i]<0){
			phi[i]=1;
		} else {
			phi[i]=0;
		}
		reinit[i]=(int)phi[i];
	}


	sedt2d(intphi,reinit,imageH,imageW);

	/*printf("ReInit @ %4d its\n",its);*/
	for(r=0;r<imageH;r++){
		for(c=0;c<imageW;c++){
			phi[r*imageW+c]=(float)intphi[r*imageW+c];
			if(phi[r*imageW+c]>0){
				phi[r*imageW+c]=0.5*sqrt(abs(phi[r*imageW+c]));
			} else {
				phi[r*imageW+c]=-0.5*sqrt(abs(phi[r*imageW+c]));
			}
			//printf("%6.3f ", phi[r*imageW+c]);
		}
		//printf("\n");
	}

	free(reinit);
	free(intphi);
}

void cuda_update(){


	dim3 dimGrid( ((imageW-1)/BLOCKDIM_X) + 1, ((imageH-1)/BLOCKDIM_Y) +1 );
	dim3 dimBlock(BLOCKDIM_X, BLOCKDIM_Y);

	cutStartTimer(ReInitTimer); // ReInit Timer Start
	updatephi<<< dimGrid, dimBlock>>>(d_phi, d_phi1, d_D,  imageW, imageH);
	hipDeviceSynchronize();
	cutStopTimer(ReInitTimer); // ReInit Timer Stop
	d_phi=d_phi1;
	


}

int main(int argc, char** argv){

	// Load the Input Image using BMPLoader
	const char *image_path = IMAGE;
	LoadBMPFile(&h_Src, &imageW, &imageH, image_path);
	D = (float *)malloc(imageW*imageH*sizeof(float));
	for(r=0;r<imageH;r++){
		for(c=0;c<imageW;c++){
			D[r*imageW+c] = h_Src[r*imageW+c].x;
		}
	}

	N = imageW*imageH;

	// Threshold based on hash defined paramters
	for(i=0;i<N;i++){
		D[i] = EPSILON - abs(D[i] - THRESHOLD);
	}

	// Init phi to SDF
	init_phi();

	// Set up CUDA Timer
	cutCreateTimer(&Timer);
	cutCreateTimer(&ReInitTimer);

	

// Allocate Memory on Device
	hipMalloc((void**)&d_D,        sizeof(float)*imageW*imageH);
	hipMalloc((void**)&d_phi,      sizeof(float)*imageW*imageH);
	hipMalloc((void**)&d_phi1,         sizeof(float)*imageW*imageH);

	// Copy Host Thresholding Data to Device Memory
	hipMemcpy(d_D, D,				sizeof(float)*imageW*imageH, hipMemcpyHostToDevice);
	hipMemcpy(d_phi1, phi, sizeof(float)*imageW*imageH, hipMemcpyHostToDevice);
	cutStartTimer(Timer);
	for(its=0;its<=ITERATIONS;its++){
		cuda_update();
		if(its%50==0)printf("Iteration %3d Total Time: %3.2f\n", its, 0.001*cutGetTimerValue(Timer));
	}

	hipMemcpy(phi, d_phi, sizeof(float)*imageW*imageH, hipMemcpyDeviceToHost);
	unsigned char *output;
	if((output = (unsigned char *) malloc(N))==NULL)printf("ME_OUTPUT\n");
	for(i=0;i<N;i++){
		if(phi[i]>0){output[i]=0;} else { output[i]=255; }
	}
	char *outputFilename= "output.raw";
	FILE *fp = fopen(outputFilename, "wb");
	size_t write = fwrite(output, 1, N, fp);
	fclose(fp);
    printf("Write '%s', %d bytes\n", outputFilename, write);

	char dummy[100];
	scanf("%c",dummy);
}